#include <iostream>
#include <cmath>
#include <memory>
#include <random>
#include <hip/hip_runtime.h>

void generateMatrix(float *matrix, int n, std::mt19937 &mt)
{
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);
    for (int i = 0; i < n; ++i)
        matrix[i] = dist(mt);
}

__global__ void compute_attention_scores_qk(
    const float *__restrict__ Q,
    const float *__restrict__ K,
    float *__restrict__ scores, // Output: [B, H, L, L]
    int B, int H, int L, int D)
{
    int q = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int b = blockIdx.z / H; // batch index
    int h = blockIdx.z % H; // head index
    if (q >= L || k >= L || b >= B)
        return;

    // Properly compute base indices considering separate b and h
    int base_q = ((b * H + h) * L + q) * D;
    int base_k = ((b * H + h) * L + k) * D;

    float score = 0.0f;
    for (int d = 0; d < D; ++d)
        score += Q[base_q + d] * K[base_k + d];

    // Properly index into scores using separate b and h
    scores[((b * H + h) * L + q) * L + k] = score / sqrtf((float)D);
}

__global__ void softmax_rows(
    float *__restrict__ scores, // in-place or out-of-place
    int B, int H, int L)
{
    int b = blockIdx.x;
    int h = blockIdx.y;
    int q = threadIdx.x + blockIdx.z * blockDim.x;

    if (q >= L)
        return;

    int row_offset = ((b * H + h) * L + q) * L;
    float max_val = scores[row_offset];

    // 1. Find max for numerical stability
    for (int k = 1; k < L; ++k)
    {
        float val = scores[row_offset + k];
        max_val = fmaxf(max_val, val);
    }

    // 2. Compute exp and sum
    float sum = 0.0f;
    for (int k = 0; k < L; ++k)
    {
        float val = expf(scores[row_offset + k] - max_val);
        scores[row_offset + k] = val;
        sum += val;
    }

    // 3. Normalize
    for (int k = 0; k < L; ++k)
    {
        scores[row_offset + k] /= sum;
    }
}

__global__ void apply_value_weights(
    const float *__restrict__ softmax, // [B, H, L, L]
    const float *__restrict__ V,       // [B, H, L, D]
    float *__restrict__ output,        // [B, H, L, D]
    int B, int H, int L, int D)
{
    int b = blockIdx.x;
    int h = blockIdx.y;
    int q = threadIdx.y + blockIdx.z * blockDim.y;

    if (q >= L)
        return;

    for (int d = threadIdx.x; d < D; d += blockDim.x)
    {
        float out = 0.0f;
        for (int k = 0; k < L; ++k)
        {
            int score_idx = ((b * H + h) * L + q) * L + k;
            int v = ((b * H + h) * L + k) * D + d;
            out += softmax[score_idx] * V[v];
        }

        int out_idx = ((b * H + h) * L + q) * D + d;
        output[out_idx] = out;
    }
}

int main()
{
    int B = 1, H = 4, L = 32, D = 64;
    
    //if (argc >= 2) L = std::atoi(argv[1]);
    //if (argc >= 3) B = std::atoi(argv[2]);
    //if (argc >= 4) H = std::atoi(argv[3]);
    //if (argc >= 5) D = std::atoi(argv[4]);

    const int size = B * H * L * D;
    const size_t bytes = size * sizeof(float);
    const int score_size = B * H * L * L;

    std::unique_ptr<float[]> h_Q(new float[size]);
    std::unique_ptr<float[]> h_K(new float[size]);
    std::unique_ptr<float[]> h_V(new float[size]);
    std::unique_ptr<float[]> h_O(new float[size]);

    // std::mt19937 mt(std::random_device{}());
    std::mt19937 mt(42);
    generateMatrix(h_Q.get(), size, mt);
    generateMatrix(h_K.get(), size, mt);
    generateMatrix(h_V.get(), size, mt);

    float *d_Q, *d_K, *d_V, *d_O, *d_scores;
    hipMalloc((void **)&d_Q, bytes);
    hipMalloc((void **)&d_K, bytes);
    hipMalloc((void **)&d_V, bytes);
    hipMalloc((void **)&d_O, bytes);
    hipMalloc(&d_scores, score_size * sizeof(float));

    hipMemcpy(d_Q, h_Q.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_K, h_K.get(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V.get(), bytes, hipMemcpyHostToDevice);

    const int WARP_SIZE = 32;
    dim3 block_1(WARP_SIZE, 8); // 256 threads per block (optimal occupancy)
    dim3 grid_1((L + block_1.x - 1) / block_1.x, (L + block_1.y - 1) / block_1.y, B * H);
    compute_attention_scores_qk<<<grid_1, block_1>>>(d_Q, d_K, d_scores, B, H, L, D);

    int T_2 = 128;
    dim3 block_2(T_2);
    dim3 grid_2(B, H, (L + T_2 - 1) / T_2);
    softmax_rows<<<grid_2, block_2>>>(d_scores, B, H, L);

    dim3 block_3(WARP_SIZE * 2, 8);
    dim3 grid_3(B, H, (L + block_3.y - 1) / block_3.y);
    apply_value_weights<<<grid_3, block_3>>>(d_scores, d_V, d_O, B, H, L, D);

    hipMemcpy(h_O.get(), d_O, bytes, hipMemcpyDeviceToHost);

    std::cout << "Sample output[0][0][0][:5]: ";
    for (int i = 0; i < 5; ++i)
        std::cout << h_O[i] << " ";
    std::cout << std::endl;

    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_O);
    hipFree(d_scores);
    return 0;
}
